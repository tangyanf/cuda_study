#include <utils.cuh>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CHECK(call) \
{\
    const hipError_t error = call;\
    if (error != hipSuccess) {\
        printf("ERROR: %s, %d\n", __FILE__, __LINE__);\
        printf("code: %d, reason: %s\n", error, hipGetErrorString(error));\
        exit(1);\
    }\
}

int main(int argc, char **argv) {
    // CudaDeviceInfo();
    if (argc != 2) {
        printf("Please select a kernel (range 0 - 11, here 0 is for NVIDIA cuBLAS).\n");
        exit(EXIT_FAILURE);
    }

    // cuda kernel num
    int kernel_num = atoi(argv[1]);
    if (kernel_num < 0 || kernel_num > 11) {
        printf("Please enter a valid kernel number (0-11).\n");
        exit(EXIT_FAILURE);
    } else {
        printf("Select kernel %d.\n", kernel_num);
    };

    // 申明句柄，创建句柄, hipblasCreate会返回一个hipblasStatus_t类型的值，用来判断句柄是否创建成功(值为0)
    hipblasHandle_t handle;
    if (hipblasCreate(&handle)) {
        printf("Create cublas handle error.\n");
        exit(EXIT_FAILURE);
    };

    // 采用cudaEvent进行gpu流计时，cudaEvent相当于在目标流中发布事件任务
    float elapsed_time;
    hipEvent_t beg, end;
    hipEventCreate(&beg);
    hipEventCreate(&end);

    // matrix size
    int size_len = 24;
    int SIZE[size_len];
    for (int i = 0; i < size_len; i++)
        SIZE[i] = 256 * (i + 1);

    int m, n, k, max_size;
    max_size = SIZE[size_len - 1];
    printf("max_size=%d\n", max_size);

    float alpha = 1.0, beta = 0.; //two arbitary input parameters，C=α*AB+β*C

    float *A = NULL, *B = NULL, *C = NULL, *C_ref = NULL;     //host matrices
    float *dA = NULL, *dB = NULL, *dC = NULL, *dC_ref = NULL; //device matrices

    A = (float *) malloc(sizeof(float) * max_size * max_size);
    B = (float *) malloc(sizeof(float) * max_size * max_size);
    C = (float *) malloc(sizeof(float) * max_size * max_size);
    C_ref = (float *) malloc(sizeof(float) * max_size * max_size);

    randomize_matrix(A, max_size * max_size);
    randomize_matrix(B, max_size * max_size);
    randomize_matrix(C, max_size * max_size);
    copy_matrix(C, C_ref, max_size * max_size);

    CHECK(hipMalloc((void **) &dA, sizeof(float) * max_size * max_size));
    CHECK(hipMalloc((void **) &dB, sizeof(float) * max_size * max_size));
    CHECK(hipMalloc((void **) &dC, sizeof(float) * max_size * max_size));
    CHECK(hipMalloc((void **) &dC_ref, sizeof(float) * max_size * max_size));

    CHECK(hipMemcpy(dA, A, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dB, B, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dC, C, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dC_ref, C_ref, sizeof(float) * max_size * max_size, hipMemcpyHostToDevice));

    int repeat_times = 10;
    for (int i = 0; i < size_len; i++) {
        m = n = k = SIZE[i];

        printf("m=n=k=%d\n", m);
        // 验证计算正确性，同时在核函数计时前预先执行一次，避免冷启动误差
        if (kernel_num != 0) {
            test_kernel(0, m, n, k, alpha, dA, dB, beta, dC_ref, handle);      // cuBLAS
            test_kernel(kernel_num, m, n, k, alpha, dA, dB, beta, dC, handle); // user define
            hipDeviceSynchronize();
            hipMemcpy(C, dC, sizeof(float) * m * n, hipMemcpyDeviceToHost);
            hipMemcpy(C_ref, dC_ref, sizeof(float) * m * n, hipMemcpyDeviceToHost);
            hipDeviceSynchronize();

            if (!verify_matrix(C_ref, C, m * n)) {
                printf("Failed to pass the correctness verification against NVIDIA cuBLAS. Exited.\n");
                exit(EXIT_FAILURE);
            }
        }
        hipDeviceSynchronize();

        hipEventRecord(beg);
        for (int j = 0; j < repeat_times; j++) {
            test_kernel(kernel_num, m, n, k, alpha, dA, dB, beta, dC, handle);
        }
        hipEventRecord(end);
        hipEventSynchronize(beg);
        hipEventSynchronize(end);
        hipEventElapsedTime(&elapsed_time, beg, end);
        elapsed_time /= 1000.; //换算成秒

        printf("Average elasped time: (%f) second, performance: (%f) GFLOPS. size: (%d).\n",
               elapsed_time / repeat_times, 2. * 1e-9 * repeat_times * m * n * k / elapsed_time, m);
        fflush(stdout);
        copy_matrix(C_ref, C, m * n); //sync C with cuBLAS to prepare for the next run
    }

    // 释放CPU和GPU空间
    free(A);
    free(B);
    free(C);
    free(C_ref);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    hipFree(dC_ref);

    return 0;
};
